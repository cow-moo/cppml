#include "hip/hip_runtime.h"
#include "backend/cuda_buffer.hpp"
#include "config.hpp"
#include <hip/hip_runtime.h>
#include <vector>

namespace backend {

using linalg::Shape;
using linalg::Strides;

constexpr int THREADS_PER_BLOCK = 256;

__constant__ void *R_BUFFER, *A_BUFFER, *B_BUFFER;
__constant__ size_t R_SHAPE[config::MAX_DIMS]; //, otherShape[config::MAX_DIMS];
__constant__ size_t R_STRIDES[config::MAX_DIMS], A_STRIDES[config::MAX_DIMS], B_STRIDES[config::MAX_DIMS];
__constant__ size_t R_OFFSET, A_OFFSET, B_OFFSET;
__constant__ size_t R_NDIM;
__constant__ size_t R_NUMEL;

template <void** Symbol>
static void write_const_symbol(const void* buffer) {
    hipMemcpyToSymbol(HIP_SYMBOL(*Symbol), &buffer, sizeof(void*));
}

template <const size_t* Symbol>
static void write_const_symbol(size_t value) {
    hipMemcpyToSymbol(HIP_SYMBOL(*Symbol), &value, sizeof(size_t));
}

template <const size_t (*Symbol)[config::MAX_DIMS]>
static void write_const_symbol(const Shape& shape) {
    hipMemcpyToSymbol(HIP_SYMBOL(*Symbol), shape.data(), config::MAX_DIMS * sizeof(size_t));
}

template <const size_t (*Symbol)[config::MAX_DIMS]>
static void write_const_symbol(const Strides& strides) {
    hipMemcpyToSymbol(HIP_SYMBOL(*Symbol), strides.data(), config::MAX_DIMS * sizeof(size_t));
}

template <size_t Input>
static void write_const_tensor(const void* buffer, const Shape& shape, const Strides& strides, size_t offset) {
    if constexpr (Input == 0) {
        write_const_symbol<&R_BUFFER>(buffer);
        write_const_symbol<&R_SHAPE>(shape);
        write_const_symbol<&R_STRIDES>(strides);
        write_const_symbol<&R_OFFSET>(offset);
        write_const_symbol<&R_NDIM>(shape.size());
        write_const_symbol<&R_NUMEL>(shape.numel());
    }
    else if constexpr (Input == 1) {
        write_const_symbol<&A_BUFFER>(buffer);
        write_const_symbol<&A_STRIDES>(strides);
        write_const_symbol<&A_OFFSET>(offset);
    }
    else {
        write_const_symbol<&B_BUFFER>(buffer);
        write_const_symbol<&B_STRIDES>(strides);
        write_const_symbol<&B_OFFSET>(offset);
    }
}

// __device__ size_t flat_to_data_idx(size_t flatIdx, std::array<size_t, config::MAX_DIMS> shape, std::array<size_t, config::MAX_DIMS> strides, size_t offset) {

// }

// Input = 0 -> rBuffer, rShape, rStrides, rOffset
// Input = 1 -> aBuffer, rShape, aStrides, aOffset
// Input = 2 -> bBuffer, rShape, bStrides, bOffset
template <size_t Input>
__device__ size_t flat_to_data_idx(size_t flatIdx) {
    size_t res = Input == 0 ? R_OFFSET :
                 Input == 1 ? A_OFFSET :
                 B_OFFSET;
    for (size_t i = R_NDIM; i-- > 0;) {
        res += (flatIdx % R_SHAPE[i]) * (
            Input == 0 ? R_STRIDES[i] :
            Input == 1 ? A_STRIDES[i] :
            B_STRIDES[i]);
        flatIdx /= R_SHAPE[i];
    }
    return res;
}

template <typename T, size_t Input>
__device__ T read_at_flat(size_t flatIdx) {
    size_t dataIdx = flat_to_data_idx<Input>(flatIdx);
    T* buffer = static_cast<T*>(Input == 0 ? R_BUFFER :
                                Input == 1 ? A_BUFFER :
                                B_BUFFER);
    return buffer[dataIdx];
}

template <typename T, size_t Input>
__device__ void write_at_flat(size_t flatIdx, T val) {
    size_t dataIdx = flat_to_data_idx<Input>(flatIdx);
    T* buffer = static_cast<T*>(Input == 0 ? R_BUFFER :
                                Input == 1 ? A_BUFFER :
                                B_BUFFER);
    buffer[dataIdx] = val;
}


template <typename T>
CudaBuffer<T>::CudaBuffer(size_t size) : DeviceBuffer<T>(BackendType::Cuda), size_(size) {
    hipMalloc(&data_, size_ * sizeof(T));
}

template <typename T>
CudaBuffer<T>::~CudaBuffer() {
    hipFree(data_);
}

template <typename T>
void CudaBuffer<T>::write_flat(const std::vector<T>& values) {
    hipMemcpy(data_, values.data(), size_ * sizeof(float), hipMemcpyHostToDevice);
}

// Vector bools perform bit packing so we must specialize
template <>
void CudaBuffer<bool>::write_flat(const std::vector<bool>& values) {
    std::vector<uint8_t> raw(values.size());
    for (size_t i = 0; i < values.size(); ++i)
        raw[i] = static_cast<uint8_t>(values[i]);
    hipMemcpy(data_, raw.data(), size_ * sizeof(uint8_t), hipMemcpyHostToDevice);
}

template <typename T>
std::vector<T> CudaBuffer<T>::read_flat() const {
    std::vector<T> res(size_);
    hipMemcpy(res.data(), data_, size_ * sizeof(float), hipMemcpyDeviceToHost);
    return res;
}

// Vector bools perform bit packing so we must specialize
template <>
std::vector<bool> CudaBuffer<bool>::read_flat() const {
    std::vector<uint8_t> raw(size_);
    hipMemcpy(raw.data(), data_, size_ * sizeof(uint8_t), hipMemcpyDeviceToHost);
    
    std::vector<bool> res(size_);
    for (size_t i = 0; i < size_; ++i) {
        res[i] = static_cast<bool>(raw[i]);
    }
    return res;
}

template <typename T>
__global__ void read_strided_kernel(T* dst) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= R_NUMEL) return;

    //size_t dataIdx = flat_to_data_idx<0>(idx);
    //dst[idx] = src[dataIdx];
    dst[idx] = read_at_flat<T, 0>(idx);
}

template <typename T>
static std::vector<T> read_strided_helper(const Shape& shape, const Strides& strides, size_t offset, T* data) {
    size_t numel = shape.numel();
    std::vector<T> res(numel);

    T* strided;
    hipMalloc(&strided, numel * sizeof(T));
    
    write_const_tensor<0>(data, shape, strides, offset);

    int blocks = (numel + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    read_strided_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(strided);
    hipMemcpy(res.data(), strided, numel * sizeof(T), hipMemcpyDeviceToHost);
    hipFree(strided);
    return res;
}

template <typename T>
std::vector<T> CudaBuffer<T>::read_strided(const Shape& shape, const Strides& strides, size_t offset) const {
    return read_strided_helper(shape, strides, offset, data_);
}

// Vector bools perform bit packing so we must specialize
template <>
std::vector<bool> CudaBuffer<bool>::read_strided(const Shape& shape, const Strides& strides, size_t offset) const {
    std::vector<uint8_t> raw = read_strided_helper<uint8_t>(shape, strides, offset, reinterpret_cast<uint8_t*>(data_));
    std::vector<bool> res(size_);
    for (size_t i = 0; i < size_; ++i)
        res[i] = static_cast<bool>(raw[i]);
    return res;
}

template <typename T>
T CudaBuffer<T>::read_at(size_t offset) const {
    T val;
    hipMemcpy(&val, &data_[offset], sizeof(T), hipMemcpyDeviceToHost);
    return val;
}

template <typename T, typename U, typename V, BinOp Op>
__global__ void apply_binary_kernel(T* temp)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= R_NUMEL) return;

    temp[idx] = 100;
    //write_at_flat<T, 0>(idx, 100);//read_at_flat<U, 1>(idx) * read_at_flat<V, 2>(idx));
}

template <typename T>
template <typename U, typename V>
void CudaBuffer<T>::apply_binary(
    const Shape& shape, const Strides& rStrides, size_t rOffset,
    DeviceBuffer<U>* a, const Strides& aStrides, size_t aOffset,
    DeviceBuffer<V>* b, const Strides& bStrides, size_t bOffset,
    BinOp op) 
{
    assert(a->backend_type() == BackendType::Cuda &&
           b->backend_type() == BackendType::Cuda);

    U* aData = static_cast<CudaBuffer<U>*>(a)->data_;
    V* bData = static_cast<CudaBuffer<V>*>(b)->data_;

    write_const_tensor<0>(data_, shape, rStrides, rOffset);
    write_const_tensor<1>(aData, shape, aStrides, aOffset);
    write_const_tensor<2>(bData, shape, bStrides, bOffset);

    int blocks = (shape.numel() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    apply_binary_kernel<T, U, V, BinOp::Add><<<blocks, THREADS_PER_BLOCK>>>(data_);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cout << "CUDA kernel error: " << hipGetErrorString(err) << std::endl;
}

template <typename T>
template <typename U, typename V>
void CudaBuffer<T>::apply_binary(
    const Shape& shape, const Strides& rStrides, size_t rOffset,
    DeviceBuffer<U>* a, const Strides& aStrides, size_t aOffset,
    V b, BinOp op)
{
    
}

template <typename T>
template <typename U>
void CudaBuffer<T>::apply_unary(
    const Shape& shape, const Strides& rStrides, size_t rOffset,
    DeviceBuffer<U>* other, const Strides& otherStrides, size_t otherOffset,
    UnOp op)
{

}

template <typename T>
void CudaBuffer<T>::reduce(
    const Shape& rShape, const Strides& rStrides, size_t rOffset,
    const DeviceBuffer<T>* other, const Strides& otherStrides, size_t otherOffset,
    const Shape& reduceShape, T identity, BinOp op) 
{

}

template <typename T>
template <typename U>
void CudaBuffer<T>::arg_reduce(
    const Shape& rShape, const Strides& rStrides, size_t rOffset,
    const DeviceBuffer<U>* other, const Strides& otherStrides, size_t otherOffset,
    size_t reduceDim, ArgRedOp op)
{

}

template <typename T>
void CudaBuffer<T>::matmul(
    const Shape& rShape, const Strides& rStrides, size_t rOffset,
    const DeviceBuffer<T>* a, const Strides& aStrides, size_t aOffset,
    const DeviceBuffer<T>* b, const Strides& bStrides, size_t bOffset,
    size_t innerDim)
{

}

template class CudaBuffer<float>;
template class CudaBuffer<unsigned long>;
template class CudaBuffer<bool>;
template class CudaBuffer<int>;
template class CudaBuffer<uint8_t>;

#define INSTANTIATE_APPLY_BINARY(T, U, V) \
template void CudaBuffer<T>::apply_binary<U, V>( \
    const Shape&, const Strides&, size_t, \
    DeviceBuffer<U>*, const Strides&, size_t, \
    DeviceBuffer<V>*, const Strides&, size_t, \
    BinOp); \
template void CudaBuffer<T>::apply_binary<U, V>( \
    const Shape&, const Strides&, size_t, \
    DeviceBuffer<U>*, const Strides&, size_t, \
    V, BinOp);

INSTANTIATE_APPLY_BINARY(float, float, float)
INSTANTIATE_APPLY_BINARY(unsigned long, unsigned long, unsigned long)
INSTANTIATE_APPLY_BINARY(int, int, int)

INSTANTIATE_APPLY_BINARY(bool, float, float)
INSTANTIATE_APPLY_BINARY(bool, unsigned long, unsigned long)
INSTANTIATE_APPLY_BINARY(bool, int, int)

#define INSTANTIATE_APPLY_UNARY(T, U) \
template void CudaBuffer<T>::apply_unary<U>( \
    const Shape&, const Strides&, size_t, \
    DeviceBuffer<U>*, const Strides&, size_t, \
    UnOp);

INSTANTIATE_APPLY_UNARY(float, float)

INSTANTIATE_APPLY_UNARY(float, bool)
INSTANTIATE_APPLY_UNARY(unsigned long, bool)
INSTANTIATE_APPLY_UNARY(int, bool)

INSTANTIATE_APPLY_UNARY(float, unsigned long)
INSTANTIATE_APPLY_UNARY(float, int)
INSTANTIATE_APPLY_UNARY(float, uint8_t)

#define INSTANTIATE_ARG_REDUCE(U) \
template void CudaBuffer<size_t>::arg_reduce<U>( \
    const Shape&, const Strides&, size_t, \
    const DeviceBuffer<U>*, const Strides&, size_t, \
    size_t, ArgRedOp);

INSTANTIATE_ARG_REDUCE(float)

}
