#include "hip/hip_runtime.h"
#include "backend/cuda_buffer.hpp"
#include "config.hpp"
#include "backend/cpu_utils.hpp"
#include <hip/hip_runtime.h>
#include <vector>
#include <bit>
#include <cstddef>
#include <limits>

namespace backend {

using linalg::Shape;
using linalg::Strides;

using ShapeArray = std::array<size_t, config::MAX_DIMS>;
using StridesArray = std::array<size_t, config::MAX_DIMS>;

constexpr size_t THREADS_PER_BLOCK = 256;
constexpr size_t MAX_REDUCE_DIM = 1024;

template <typename T>
static constexpr T ceil_div(T a, T b) {
    static_assert(std::is_integral_v<T>, "ceil_div requires integral types");
    return (a + b - 1) / b;
}

constexpr size_t bit_ceil(size_t x) {
    if (x <= 1) return 1;

    // Find the position of the highest bit set
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
#if SIZE_MAX > 0xFFFFFFFF
    x |= x >> 32; // Needed for 64-bit size_t
#endif
    return x + 1;
}

__device__ size_t flat_to_data_idx(
    size_t flatIdx, 
    const size_t ndim, 
    const ShapeArray shape,
    const StridesArray strides,
    const size_t offset) 
{
    size_t res = offset;
    for (size_t i = ndim; i-- > 0;) {
        res += (flatIdx % shape[i]) * strides[i];
        flatIdx /= shape[i];
    }
    return res;
}

template <typename T>
CudaBuffer<T>::CudaBuffer(size_t size) : DeviceBuffer<T>(BackendType::Cuda), size_(size) {
    hipMalloc(&data_, size_ * sizeof(T));
}

template <typename T>
CudaBuffer<T>::~CudaBuffer() {
    hipFree(data_);
}

template <typename T>
void CudaBuffer<T>::write_flat(const std::vector<T>& values) {
    //std::cout << size_ << " " << values.size() << std::endl;
    assert(size_ == values.size());
    hipMemcpy(data_, values.data(), size_ * sizeof(T), hipMemcpyHostToDevice);
}

// Vector bools perform bit packing so we must specialize
template <>
void CudaBuffer<bool>::write_flat(const std::vector<bool>& values) {
    assert(sizeof(bool) == sizeof(uint8_t));
    //std::cout << size_ << " " << values.size() << std::endl;
    assert(size_ == values.size());
    std::vector<uint8_t> raw(values.size());
    for (size_t i = 0; i < values.size(); ++i)
        raw[i] = static_cast<uint8_t>(values[i]);
    hipMemcpy(data_, raw.data(), size_ * sizeof(bool), hipMemcpyHostToDevice);
}

template <typename T>
std::vector<T> CudaBuffer<T>::read_flat() const {
    std::vector<T> res(size_);
    hipMemcpy(res.data(), data_, size_ * sizeof(T), hipMemcpyDeviceToHost);
    return res;
}

// Vector bools perform bit packing so we must specialize
template <>
std::vector<bool> CudaBuffer<bool>::read_flat() const {
    std::vector<uint8_t> raw(size_);
    hipMemcpy(raw.data(), data_, size_ * sizeof(uint8_t), hipMemcpyDeviceToHost);
    
    std::vector<bool> res(size_);
    for (size_t i = 0; i < size_; ++i) {
        res[i] = static_cast<bool>(raw[i]);
    }
    return res;
}

template <typename T>
__global__ void read_strided_kernel(
    T* dst, 
    const T* src,
    const size_t numel,
    const size_t ndim, 
    const ShapeArray shape,
    const StridesArray strides,
    const size_t offset) 
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numel) return;

    dst[idx] = src[flat_to_data_idx(idx, ndim, shape, strides, offset)];
}

template <typename T>
static std::vector<T> read_strided_helper(const Shape& shape, const Strides& strides, size_t offset, T* data) {
    size_t numel = shape.numel();
    std::vector<T> res(numel);

    T* strided;
    hipMalloc(&strided, numel * sizeof(T));
    
    int blocks = ceil_div(numel, THREADS_PER_BLOCK);
    read_strided_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(
        strided,
        data,
        numel,
        shape.size(),
        shape.array(),
        strides.array(),
        offset
    );
    hipMemcpy(res.data(), strided, numel * sizeof(T), hipMemcpyDeviceToHost);
    hipFree(strided);
    return res;
}

template <typename T>
std::vector<T> CudaBuffer<T>::read_strided(const Shape& shape, const Strides& strides, size_t offset) const {
    return read_strided_helper(shape, strides, offset, data_);
}

// Vector bools perform bit packing so we must specialize
template <>
std::vector<bool> CudaBuffer<bool>::read_strided(const Shape& shape, const Strides& strides, size_t offset) const {
    std::vector<uint8_t> raw = read_strided_helper<uint8_t>(shape, strides, offset, reinterpret_cast<uint8_t*>(data_));
    std::vector<bool> res(shape.numel());
    for (size_t i = 0; i < raw.size(); ++i)
       res[i] = static_cast<bool>(raw[i]);
    return res;
}

template <typename T>
T CudaBuffer<T>::read_at(size_t offset) const {
    T val;
    hipMemcpy(&val, &data_[offset], sizeof(T), hipMemcpyDeviceToHost);
    return val;
}

template <typename T, typename U, typename V, size_t Op>
__global__ void apply_binary_kernel(
    size_t numel, size_t ndim, const ShapeArray shape,
    T* rData, const StridesArray rStrides, const size_t rOffset,
    U* aData, const StridesArray aStrides, const size_t aOffset,
    V* bData, const StridesArray bStrides, const size_t bOffset)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numel) return;

    size_t rIdx = flat_to_data_idx(idx, ndim, shape, rStrides, rOffset);
    size_t aIdx = flat_to_data_idx(idx, ndim, shape, aStrides, aOffset);
    size_t bIdx = flat_to_data_idx(idx, ndim, shape, bStrides, bOffset);

    constexpr auto fn = cpu_utils::binop_table<T, U, V>[Op];
    rData[rIdx] = fn(aData[aIdx], bData[bIdx]);
}

template <typename T>
template <typename U, typename V>
void CudaBuffer<T>::apply_binary(
    const Shape& shape, const Strides& rStrides, size_t rOffset,
    DeviceBuffer<U>* a, const Strides& aStrides, size_t aOffset,
    DeviceBuffer<V>* b, const Strides& bStrides, size_t bOffset,
    BinOp op) 
{
    using Kernel = void(*)(size_t, size_t, const ShapeArray,
                          T*, const StridesArray, const size_t,
                          U*, const StridesArray, const size_t,
                          V*, const StridesArray, const size_t);
    static constexpr auto lambda = []<size_t Op>() -> Kernel {
        return [](size_t numel, size_t ndim, const ShapeArray shape,
                  T* rData, const StridesArray rStrides, const size_t rOffset,
                  U* aData, const StridesArray aStrides, const size_t aOffset,
                  V* bData, const StridesArray bStrides, const size_t bOffset) {
            int blocks = ceil_div(numel, THREADS_PER_BLOCK);
            apply_binary_kernel<T, U, V, Op><<<blocks, THREADS_PER_BLOCK>>>(
                numel, ndim, shape,
                rData, rStrides, rOffset,
                aData, aStrides, aOffset,
                bData, bStrides, bOffset
            );
        };
    };
    static constexpr auto table = cpu_utils::make_kernel_table<BinOp>(lambda);

    assert(a->backend_type() == BackendType::Cuda &&
           b->backend_type() == BackendType::Cuda);

    U* aData = static_cast<CudaBuffer<U>*>(a)->data_;
    V* bData = static_cast<CudaBuffer<V>*>(b)->data_;

    table[static_cast<size_t>(op)](
        shape.numel(), shape.size(), shape.array(),
        data_, rStrides.array(), rOffset,
        aData, aStrides.array(), aOffset,
        bData, bStrides.array(), bOffset
    );
}

template <typename T, typename U, typename V, size_t Op>
__global__ void apply_binary_kernel(
    size_t numel, size_t ndim, const ShapeArray shape,
    T* rData, const StridesArray rStrides, const size_t rOffset,
    U* aData, const StridesArray aStrides, const size_t aOffset,
    V b)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numel) return;

    size_t rIdx = flat_to_data_idx(idx, ndim, shape, rStrides, rOffset);
    size_t aIdx = flat_to_data_idx(idx, ndim, shape, aStrides, aOffset);

    constexpr auto fn = cpu_utils::binop_table<T, U, V>[Op];
    rData[rIdx] = fn(aData[aIdx], b);
}

template <typename T>
template <typename U, typename V>
void CudaBuffer<T>::apply_binary(
    const Shape& shape, const Strides& rStrides, size_t rOffset,
    DeviceBuffer<U>* a, const Strides& aStrides, size_t aOffset,
    V b, BinOp op)
{
    using Kernel = void(*)(size_t, size_t, const ShapeArray,
                          T*, const StridesArray, const size_t,
                          U*, const StridesArray, const size_t,
                          V);
    static constexpr auto lambda = []<size_t Op>() -> Kernel {
        return [](size_t numel, size_t ndim, const ShapeArray shape,
                  T* rData, const StridesArray rStrides, const size_t rOffset,
                  U* aData, const StridesArray aStrides, const size_t aOffset,
                  V b) {
            int blocks = (numel + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            apply_binary_kernel<T, U, V, Op><<<blocks, THREADS_PER_BLOCK>>>(
                numel, ndim, shape,
                rData, rStrides, rOffset,
                aData, aStrides, aOffset,
                b
            );
        };
    };
    static constexpr auto table = cpu_utils::make_kernel_table<BinOp>(lambda);

    assert(a->backend_type() == BackendType::Cuda);

    U* aData = static_cast<CudaBuffer<U>*>(a)->data_;

    table[static_cast<size_t>(op)](
        shape.numel(), shape.size(), shape.array(),
        data_, rStrides.array(), rOffset,
        aData, aStrides.array(), aOffset,
        b
    );
}

template <typename T, typename U, size_t Op>
__global__ void apply_unary_kernel(
    size_t numel, size_t ndim, const ShapeArray shape,
    T* rData, const StridesArray rStrides, const size_t rOffset,
    U* aData, const StridesArray aStrides, const size_t aOffset)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numel) return;

    size_t rIdx = flat_to_data_idx(idx, ndim, shape, rStrides, rOffset);
    size_t aIdx = flat_to_data_idx(idx, ndim, shape, aStrides, aOffset);

    constexpr auto fn = cpu_utils::unop_table<T, U>[Op];
    rData[rIdx] = fn(aData[aIdx]);
}

template <typename T>
template <typename U>
void CudaBuffer<T>::apply_unary(
    const Shape& shape, const Strides& rStrides, size_t rOffset,
    DeviceBuffer<U>* other, const Strides& otherStrides, size_t otherOffset,
    UnOp op)
{
    using Kernel = void(*)(size_t, size_t, const ShapeArray,
                          T*, const StridesArray, const size_t,
                          U*, const StridesArray, const size_t);
    static constexpr auto lambda = []<size_t Op>() -> Kernel {
        return [](size_t numel, size_t ndim, const ShapeArray shape,
                  T* rData, const StridesArray rStrides, const size_t rOffset,
                  U* otherData, const StridesArray otherStrides, const size_t otherOffset) {
            int blocks = (numel + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            apply_unary_kernel<T, U, Op><<<blocks, THREADS_PER_BLOCK>>>(
                numel, ndim, shape,
                rData, rStrides, rOffset,
                otherData, otherStrides, otherOffset
            );
        };
    };
    static constexpr auto table = cpu_utils::make_kernel_table<UnOp>(lambda);

    assert(other->backend_type() == BackendType::Cuda);

    U* otherData = static_cast<CudaBuffer<U>*>(other)->data_;

    table[static_cast<size_t>(op)](
        shape.numel(), shape.size(), shape.array(),
        data_, rStrides.array(), rOffset,
        otherData, otherStrides.array(), otherOffset
    );
}

template <typename T, size_t Op>
__global__ void reduce_kernel_final(
    const size_t rNdim, const ShapeArray rShape, 
    T* rData, const StridesArray rStrides, const size_t rOffset,
    const size_t reduceDim, const size_t otherNdim, const ShapeArray otherShape, 
    T* otherData, const StridesArray otherStrides, const size_t otherOffset)
{
    __shared__ T sdata[MAX_REDUCE_DIM];

    assert(reduceDim <= MAX_REDUCE_DIM);
    assert(blockDim.x >= reduceDim);

    size_t tid = threadIdx.x;

    if (threadIdx.x < reduceDim) {
        size_t otherFlatIdx = blockIdx.x * reduceDim + threadIdx.x;
        size_t otherDataIdx = flat_to_data_idx(otherFlatIdx, otherNdim, otherShape, otherStrides, otherOffset);
        sdata[tid] = otherData[otherDataIdx];
    }
    __syncthreads();

    constexpr auto fn = cpu_utils::binop_table<T, T, T>[Op];
    for (size_t s = blockDim.x >> 1; s > 0; s >>= 1) {
        if (tid < s && tid + s < reduceDim)
            sdata[tid] = fn(sdata[tid], sdata[tid + s]);
        __syncthreads();
    }

    if (tid == 0) {
        size_t rDataIdx = flat_to_data_idx(blockIdx.x, rNdim, rShape, rStrides, rOffset);
        rData[rDataIdx] = sdata[0];
    }
}

// Owns newData and assumes flat (default strides, 0 offset)
// newShape[-1] is new reduce dim
// curData guaranteed to share a prefix with newShape[:-1]
template <typename T, size_t Op>
__global__ void reduce_kernel(
    T* newData, const size_t newReduceDim,
    const T* curData, const size_t curNdim,
    const ShapeArray curShape, const StridesArray curStrides, const size_t curOffset,
    const size_t curReduceDim)
{
    // Trying to use extern dynamic sizing leads to some symbol collision
    // from the explicit instantiation of CudaBuffer
    __shared__ T sdata[MAX_REDUCE_DIM];

    size_t tid = threadIdx.x;
    size_t bid = blockIdx.x;
    size_t bdim = blockDim.x;

    size_t finalIdx = bid / newReduceDim;
    size_t intermediateIdx = bid % newReduceDim;
    // We can describe any position in newData (finalIdx, intermediateIdx)
    // For each of these we want to reduce starting from curData (finalIdx, intermediateIdx * bdim)
    // Up to bdim elements or until we hit a reduction boundary
    // Each finalIdx corresponds to an index in final reduction
    assert(curReduceDim > intermediateIdx * bdim);
    size_t blockReduceDim = min(curReduceDim - intermediateIdx * bdim, bdim);
    if (tid < blockReduceDim) {
        size_t curFlatIdx = finalIdx * curReduceDim + intermediateIdx * bdim + tid;
        size_t curDataIdx = flat_to_data_idx(curFlatIdx, curNdim, curShape, curStrides, curOffset);
        sdata[tid] = curData[curDataIdx];
    }
    __syncthreads();

    constexpr auto fn = cpu_utils::binop_table<T, T, T>[Op];
    for (size_t s = bdim >> 1; s > 0; s >>= 1) {
        if (tid < s && tid + s < blockReduceDim)
            sdata[tid] = fn(sdata[tid], sdata[tid + s]);
        __syncthreads();
    }

    if (tid == 0) {
        newData[bid] = sdata[0];
    }
}

template <typename T>
void CudaBuffer<T>::reduce(
    const Shape& rShape, const Strides& rStrides, size_t rOffset,
    const DeviceBuffer<T>* other, const Strides& otherStrides, size_t otherOffset,
    const Shape& reduceShape, T identity, BinOp op) 
{
    using Kernel = void(*)(T*, const size_t,
                           const T*, const size_t,
                           const ShapeArray, const StridesArray, const size_t,
                           const size_t);
    static constexpr auto lambda = []<size_t Op>() -> Kernel {
        return [](T* newData, const size_t newReduceDim,
                  const T* curData, const size_t curNdim,
                  const ShapeArray curShape, const StridesArray curStrides, const size_t curOffset,
                  const size_t curReduceDim) {
            size_t newSize = newReduceDim;
            for (size_t i = 0; i < curNdim; ++i)
                newSize *= curShape[i];
            newSize /= curReduceDim;

            size_t bdim = bit_ceil(ceil_div(curReduceDim, newReduceDim));
            assert(bdim <= MAX_REDUCE_DIM);
            // newReduceDim != 1 implies bdim == MAX_REDUCE_DIM
            assert(newReduceDim == 1 || bdim == MAX_REDUCE_DIM);

            reduce_kernel<T, Op><<<newSize, bdim>>>(
                newData, newReduceDim,
                curData, curNdim,
                curShape, curStrides, curOffset,
                curReduceDim
            );
        };
    };
    static constexpr auto table = cpu_utils::make_kernel_table<BinOp>(lambda);

    assert(other->backend_type() == BackendType::Cuda);

    T* otherData = static_cast<const CudaBuffer*>(other)->data_;
    size_t rNumel = rShape.numel();

    // Get full shape of other
    Shape curShape(rShape);
    for (auto dim : reduceShape)
        curShape.push_back(dim);

    size_t curNdim = curShape.size();
    T* curData = otherData;
    Strides curStrides = otherStrides;
    size_t curOffset = otherOffset;
    // Remaining dimension to reduce for intermediate sum
    size_t curReduceDim = reduceShape.numel();

    // Do intermediate reductions until remaining dimension to reduce fits in chunk size
    while (curReduceDim > MAX_REDUCE_DIM) {
        // Dimension to reduce to (>1 due to comparison in while loop)
        size_t newReduceDim = ceil_div(curReduceDim, MAX_REDUCE_DIM);

        // Intermediate sum allocation
        size_t newSize = rNumel * newReduceDim;
        T* newData;
        hipMalloc(&newData, newSize * sizeof(T));

        // Dispatch kernel
        table[static_cast<size_t>(op)](
            newData, newReduceDim,
            curData, curNdim,
            curShape.array(), curStrides.array(), curOffset,
            curReduceDim
        );

        // Free curData if it was allocated
        if (curData != otherData)
            hipFree(curData);
        
        // Update cur
        curNdim = rShape.size() + 1;
        curShape = rShape;
        curShape.push_back(newReduceDim);
        curData = newData;
        curStrides = Strides(curShape);
        curOffset = 0;
        curReduceDim = newReduceDim;
    }

    table[static_cast<size_t>(op)](
        data_, 1,
        curData, curNdim,
        curShape.array(), curStrides.array(), curOffset,
        curReduceDim
    );

    if (curData != otherData)
        hipFree(curData);
}

// Owns newData and assumes flat (default strides, 0 offset)
// newShape[-1] is new reduce dim
// curData guaranteed to share a prefix with newShape[:-1]
// if curIdxs == nullptr then assume default strides
// if newData == nullptr then don't write
template <typename T, size_t Op>
__global__ void arg_reduce_kernel(
    T* newData, size_t* newIdxs, const size_t newReduceDim,
    const T* curData, const size_t* curIdxs, const size_t curNdim,
    const ShapeArray curShape, const StridesArray curStrides, const size_t curOffset,
    const size_t curReduceDim)
{
    // Trying to use extern dynamic sizing leads to some symbol collision
    // from the explicit instantiation of CudaBuffer
    __shared__ T sdata[MAX_REDUCE_DIM];
    __shared__ size_t sidxs[MAX_REDUCE_DIM];

    size_t tid = threadIdx.x;
    size_t bid = blockIdx.x;
    size_t bdim = blockDim.x;

    size_t finalIdx = bid / newReduceDim;
    size_t intermediateIdx = bid % newReduceDim;
    // We can describe any position in newData (finalIdx, intermediateIdx)
    // For each of these we want to reduce starting from curData (finalIdx, intermediateIdx * bdim)
    // Up to bdim elements or until we hit a reduction boundary
    // Each finalIdx corresponds to an index in final reduction
    assert(curReduceDim > intermediateIdx * bdim);
    size_t blockReduceDim = min(curReduceDim - intermediateIdx * bdim, bdim);
    if (tid < blockReduceDim) {
        size_t curFlatIdx = finalIdx * curReduceDim + intermediateIdx * bdim + tid;
        size_t curDataIdx = flat_to_data_idx(curFlatIdx, curNdim, curShape, curStrides, curOffset);
        sdata[tid] = curData[curDataIdx];
        sidxs[tid] = curIdxs == nullptr ? intermediateIdx * bdim + tid : curIdxs[curDataIdx];
    }
    __syncthreads();

    for (size_t s = bdim >> 1; s > 0; s >>= 1) {
        if (tid < s && tid + s < blockReduceDim) {
            if constexpr (Op == static_cast<size_t>(ArgRedOp::Max)) {
                if (sdata[tid] < sdata[tid + s]) {
                    sdata[tid] = sdata[tid + s];
                    sidxs[tid] = sidxs[tid + s];
                }
            }
            else {
                if (sdata[tid] > sdata[tid + s]) {
                    sdata[tid] = sdata[tid + s];
                    sidxs[tid] = sidxs[tid + s];
                }
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        if (newData != nullptr)
            newData[bid] = sdata[0];
        newIdxs[bid] = sidxs[0];
    }
}

template <typename T>
template <typename U>
void CudaBuffer<T>::arg_reduce(
    const Shape& rShape, const Strides& rStrides, size_t rOffset,
    const DeviceBuffer<U>* other, const Strides& otherStrides, size_t otherOffset,
    size_t reduceDim, ArgRedOp op)
{
    static_assert(std::is_same_v<T, size_t>, "arg_reduce only works with T = size_t");

    using Kernel = void(*)(U*, size_t*, const size_t,
                           const U*, const size_t*, const size_t,
                           const ShapeArray, const StridesArray, const size_t,
                           const size_t);
    static constexpr auto lambda = []<size_t Op>() -> Kernel {
        return [](U* newData, size_t* newIdxs, const size_t newReduceDim,
                  const U* curData, const size_t* curIdxs, const size_t curNdim,
                  const ShapeArray curShape, const StridesArray curStrides, const size_t curOffset,
                  const size_t curReduceDim) {
            size_t newSize = newReduceDim;
            for (size_t i = 0; i < curNdim; ++i)
                newSize *= curShape[i];
            newSize /= curReduceDim;

            size_t bdim = bit_ceil(ceil_div(curReduceDim, newReduceDim));
            assert(bdim <= MAX_REDUCE_DIM);
            // newReduceDim != 1 implies bdim == MAX_REDUCE_DIM
            assert(newReduceDim == 1 || bdim == MAX_REDUCE_DIM);

            arg_reduce_kernel<U, Op><<<newSize, bdim>>>(
                newData, newIdxs, newReduceDim,
                curData, curIdxs, curNdim,
                curShape, curStrides, curOffset,
                curReduceDim
            );
        };
    };
    static constexpr auto table = cpu_utils::make_kernel_table<ArgRedOp>(lambda);

    assert(other->backend_type() == BackendType::Cuda);

    U* otherData = static_cast<const CudaBuffer<U>*>(other)->data_;
    size_t rNumel = rShape.numel();

    // Get full shape of other
    Shape curShape(rShape);
    curShape.push_back(reduceDim);

    size_t curNdim = curShape.size();
    U* curData = otherData;
    size_t* curIdxs = nullptr;
    Strides curStrides = otherStrides;
    size_t curOffset = otherOffset;
    // Remaining dimension to reduce for intermediate sum
    size_t curReduceDim = reduceDim;

    // Do intermediate reductions until remaining dimension to reduce fits in chunk size
    while (curReduceDim > MAX_REDUCE_DIM) {
        // Dimension to reduce to (>1 due to comparison in while loop)
        size_t newReduceDim = ceil_div(curReduceDim, MAX_REDUCE_DIM);

        // Intermediate sum allocation
        size_t newSize = rNumel * newReduceDim;
        U* newData;
        size_t* newIdxs;
        hipMalloc(&newData, newSize * sizeof(U));
        hipMalloc(&newIdxs, newSize * sizeof(size_t));

        // Dispatch kernel
        table[static_cast<size_t>(op)](
            newData, newIdxs, newReduceDim,
            curData, curIdxs, curNdim,
            curShape.array(), curStrides.array(), curOffset,
            curReduceDim
        );

        // Free curData if it was allocated
        if (curData != otherData) {
            hipFree(curData);
            hipFree(curIdxs);
        }
        
        // Update cur
        curNdim = rShape.size() + 1;
        curShape = rShape;
        curShape.push_back(newReduceDim);
        curData = newData;
        curIdxs = newIdxs;
        curStrides = Strides(curShape);
        curOffset = 0;
        curReduceDim = newReduceDim;
    }

    table[static_cast<size_t>(op)](
        nullptr, data_, 1,
        curData, curIdxs, curNdim,
        curShape.array(), curStrides.array(), curOffset,
        curReduceDim
    );

    if (curData != otherData) {
        hipFree(curData);
        hipFree(curIdxs);
    }
}

constexpr size_t BM = 64;
constexpr size_t BN = 64;
constexpr size_t BK = 32;

constexpr size_t TM = 4;
constexpr size_t TN = 4;

constexpr size_t NTHREADS = BM * BN / TM / TN;

static_assert(NTHREADS % BK == 0);
constexpr size_t A_STRIDE_INNER = NTHREADS / BK;
static_assert(NTHREADS % BN == 0);
constexpr size_t B_STRIDE_INNER = NTHREADS / BN;

// BS * M * K
// BS * K * N
// BS * M * N
// BS * ceil_div(M, TM) * ceil_div(N, TN)
// BS * ceil_div(M, BM) * ceil_div(N, BN) blocks
// 1 * (BM / TM) * (BN / TN) threads per block
// z, x, y
template <typename T>
__global__ void matmul_kernel(
    const size_t ndim, const ShapeArray batchShape, 
    const size_t m, const size_t n, const size_t k,
    T* rData, const StridesArray rStrides, const size_t rOffset,
    const T* aData, const StridesArray aStrides, const size_t aOffset,
    const T* bData, const StridesArray bStrides, const size_t bOffset)
{
    __shared__ T sa[BM * BK];
    __shared__ T sb[BK * BN];

    size_t batchIdx = blockIdx.z;
    size_t rBatchOffset = flat_to_data_idx(batchIdx, ndim, batchShape, rStrides, rOffset);
    size_t aBatchOffset = flat_to_data_idx(batchIdx, ndim, batchShape, aStrides, aOffset);
    size_t bBatchOffset = flat_to_data_idx(batchIdx, ndim, batchShape, bStrides, bOffset);

    size_t tid = threadIdx.x + threadIdx.y * blockDim.x;

    size_t aIdxM = blockIdx.x * BM + tid / BK;
    size_t aIdxK = tid % BK;

    size_t bIdxK = tid / BN;
    size_t bIdxN = blockIdx.y * BN + tid % BN;

    size_t rIdxM = threadIdx.x * TM + blockIdx.x * BM;
    size_t rIdxN = threadIdx.y * TN + blockIdx.y * BN;

    size_t tm = min(TM, m > rIdxM ? m - rIdxM : 0);
    size_t tn = min(TN, n > rIdxN ? n - rIdxN : 0);

    T res[TM * TN] = {0};
    T aReg[TM], bReg[TN];

    // Outer loop
    for (size_t kIdx = 0; kIdx < k; kIdx += BK) {
        // Load SMEM
        if (aIdxK < k) {
            for (size_t i = 0; i < BM * BK / NTHREADS; ++i) {
                if (aIdxM + i * A_STRIDE_INNER < m) {
                    sa[i * NTHREADS + tid] = 
                        aData[aBatchOffset + 
                              (aIdxM + i * A_STRIDE_INNER) * aStrides[ndim] +
                              aIdxK * aStrides[ndim + 1]];
                }
            }
        }

        if (bIdxN < n) {
            for (size_t i = 0; i < BK * BN / NTHREADS; ++i) {
                if (bIdxK + i * B_STRIDE_INNER < k) {
                    sb[i * NTHREADS + tid] =
                        bData[bBatchOffset +
                              (bIdxK + i * B_STRIDE_INNER) * bStrides[ndim] +
                              bIdxN * bStrides[ndim + 1]];
                }
            }
        }

        __syncthreads();

        aIdxK += BK;
        bIdxK += BK;

        // Inner loop
        for (size_t i = 0; i < min(BK, k - kIdx); ++i) {
            // Load regs
            for (size_t j = 0; j < tm; ++j) {
                aReg[j] = sa[i + (j + threadIdx.x * TM) * BK];
            }

            for (size_t k = 0; k < tn; ++k) {
                bReg[k] = sb[k + threadIdx.y * TN + i * BN];
            }

            // Outer product
            for (size_t j = 0; j < tm; ++j) {
                for (size_t k = 0; k < tn; ++k) {
                    res[k + j * TN] += aReg[j] * bReg[k];
                }
            }
        }

        __syncthreads();
    }

    size_t rThreadOffset = rBatchOffset + rIdxM * rStrides[ndim] + rIdxN * rStrides[ndim + 1];

    // Write results
    for (size_t j = 0; j < tm; ++j) {
        for (size_t k = 0; k < tn; ++k) {
            rData[rThreadOffset + j * rStrides[ndim] + k * rStrides[ndim + 1]] = res[k + j * TN];
        }
    }
}

template <typename T>
void CudaBuffer<T>::matmul(
    const Shape& rShape, const Strides& rStrides, size_t rOffset,
    const DeviceBuffer<T>* a, const Strides& aStrides, size_t aOffset,
    const DeviceBuffer<T>* b, const Strides& bStrides, size_t bOffset,
    size_t innerDim)
{
    Shape batchShape = rShape;
    batchShape.pop_back(); batchShape.pop_back();
    size_t m = rShape[-2];
    size_t n = rShape[-1];

    assert(a->backend_type() == BackendType::Cuda &&
           b->backend_type() == BackendType::Cuda);

    const T* aData = static_cast<const CudaBuffer*>(a)->data_;
    const T* bData = static_cast<const CudaBuffer*>(b)->data_;

    constexpr dim3 blockDim(BM / TM, BN / TN, 1);
    dim3 gridDim(ceil_div(m, BM), ceil_div(n, BN), batchShape.numel());
    matmul_kernel<T><<<gridDim, blockDim>>>(
        batchShape.size(), batchShape.array(),
        m, n, innerDim,
        data_, rStrides.array(), rOffset,
        aData, aStrides.array(), aOffset,
        bData, bStrides.array(), bOffset
    );
}

#include "cuda_buffer_inst.inc"

}

    // size_t aBlockOffset = blockDim.x * BM * aStrides[ndim];
    // size_t bBlockOffset = blockDim.y * BN * bStrides[ndim + 1];

    // size_t aThreadOffset = tid / BK * aStrides[ndim] + tid % BK * aStrides[ndim + 1];
    // size_t bThreadOffset = tid / BN * bStrides[ndim] + tid % BN * bStrides[ndim + 1];

    // size_t aStrideOuter = BK * aStrides[ndim + 1];
    // size_t aStrideInner = NTHREADS / BK * aStrides[ndim];
    // static_assert(NTHREADS % BK == 0);

    // size_t bStrideOuter = BK * bStrides[ndim];
    // size_t bStrideInner = NTHREADS / BN * bStrides[ndim];
    // static_assert(NTHREADS % BN == 0);

    // size_t aIdx = aBatchOffset + aBlockOffset + aThreadOffset;
    // size_t bIdx = bBatchOffset + bBlockOffset + bThreadOffset;